// Name: Rishita Meharishi
//Date: May 13,2025
// Course CSC656-01 Spring 2025
// Citations:NVDIA developer blog, github copilot to fix errors, class slides, 
//modifiled the given code of my own .

// step 5

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
    // Compute the thread's unique index and stride
    int index = threadIdx.x;
    int stride = blockDim.x;

    // Each thread processes a subset of the array
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 29; // Size of the arrays (1 million elements)

    // Allocate memory for arrays using Unified Memory
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Launch kernel with 256 threads and 1 block
    int threads_per_block = 256;
    int blocks_per_grid = 1;

    // from the NVIDIA tutotial article
    add<<<1, 256>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print a few results to verify correctness
    std::cout << "y[0] = " << y[0] << ", y[N-1] = " << y[N-1] << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}