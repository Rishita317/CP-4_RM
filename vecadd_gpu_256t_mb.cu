// step 7


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
    // Compute the thread's unique index and stride
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Each thread processes a subset of the array
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 20; // Size of the arrays (1 million elements)

    // Allocate memory for arrays using Unified Memory
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Define block size and calculate the number of blocks
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Print the number of thread blocks
    std::cout << "Number of thread blocks: " << numBlocks << std::endl;

    // Launch kernel with calculated number of blocks and threads per block
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print a few results to verify correctness
    std::cout << "y[0] = " << y[0] << ", y[N-1] = " << y[N-1] << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}