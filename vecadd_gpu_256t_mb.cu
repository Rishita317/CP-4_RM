#include "hip/hip_runtime.h"
// Name: Rishita Meharishi
//Date: May 13,2025
// Course CSC656-01 Spring 2025
// Citations:NVDIA developer blog, github copilot to fix errors, class slides, 
//modifiled the given code of my own .

// step 8
#include <iostream>
#include <math.h>

// CUDA kernel to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
    // Compute the thread's unique index and stride
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Each thread processes a subset of the array
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 29; // Size of the arrays (1 million elements)

    // Allocate memory for arrays using Unified Memory
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Define block size and calculate the number of blocks
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Print the number of thread blocks
    std::cout << "Number of thread blocks: " << numBlocks << std::endl;

    // Launch kernel with calculated number of blocks and threads per block
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    // adding varification

    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y[i]-3.0f));
    
    // Print a few results to verify correctness
    std::cout << "y[0] = " << y[0] << ", y[N-1] = " << y[N-1] << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}