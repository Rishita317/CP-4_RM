// Name: Rishita Meharishi
//Date: May 13,2025
// Course CSC656-01 Spring 2025
// Citations:NVDIA developer blog, github copilot to fix errors, class slides, 
//modifiled the given code of my own .


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to add the elements of two arrays
// TODO done: Updating the add function with a prefix __global__

__global__ void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    // Updated problem size to 29 instead of 21
    int N = 1 << 29;

    // Allocate memory for arrays using Unified Memory
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Launch the add kernel on the GPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Free the allocated memory
    hipFree(x);
    hipFree(y);

    return 0;
}